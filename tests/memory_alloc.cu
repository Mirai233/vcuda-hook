#include <hip/hip_runtime.h>
#include <iostream>

static void checkCuda(hipError_t result, const char *message) {
    if (result != hipSuccess) {
        std::cerr << message << ": " << hipGetErrorString(result) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

int main() {
    constexpr size_t bytes = 256 * 1024 * 1024;  // allocate 256MB
    void *device_ptr = nullptr;

    checkCuda(hipMalloc(&device_ptr, bytes), "hipMalloc failed");
    std::cout << "Allocated " << bytes << " bytes on GPU at address " << device_ptr << std::endl;

    checkCuda(hipFree(device_ptr), "hipFree failed");
    std::cout << "Freed device memory" << std::endl;

    return 0;
}
