
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// CUDA kernel to add elements of two arrays
__global__
void add(int n, float* x, float* y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

void testTraditionalCudaMemory(int N) {
    float* h_x = new float[N];
    float* h_y = new float[N];

    float* d_x;
    float* d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    // init data
    for (int i = 0; i < N; i++) {
        h_x[i] = 1.0f;
        h_y[i] = 2.0f;
    }

    // int blockSize = 256;
    // int numBlocks = (N + blockSize - 1) / blockSize;

    hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, N * sizeof(float), hipMemcpyHostToDevice);

    // Modify the kernel to use shared memory
    for (int i = 0; i < 10; i++) {
        // Define shared memory size (per block)
        // size_t sharedMemSize = blockSize * sizeof(float);

        // Launch kernel with shared memory
        // add<<<numBlocks, blockSize, sharedMemSize>>>(N, d_x, d_y);
        dim3 gridDim(8, 256, 1);
        dim3 blockDim(256, 1, 1);
        size_t sharedMemBytes = 16384;  // 16KB
        add<<<gridDim, blockDim, sharedMemBytes>>>(N, d_x, d_y);

        hipDeviceSynchronize();
    }
    hipMemcpy(h_y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(h_y[i] - 3.0f));
    std::cout << "Max : " << maxError << std::endl;

    hipFree(d_x);
    hipFree(d_y);

    delete[] h_x;
    delete[] h_y;
}

int main(void) {
    int N = 1 << 28;

    std::cout << "Test Data Size: " << (N * sizeof(float) / (1024 * 1024)) << " MB" << std::endl;

    testTraditionalCudaMemory(N);

    return 0;
}
